#include "hip/hip_runtime.h"
#include "Matrix.cuh"

#pragma region ACTIVATION_FUNCTIONS

__device__ float sigmoid(float net) {
	return 1.0f / (1.0f + exp(-net));
}

__device__ float sigmoidDerivative(float out) {
	return out * (1 - out);
}

__device__ float step(float net) {
	if (net > 0.5) return 1;
	else return 0;
}

__device__ float stepDerivative(float out) {
	return 1;
}

__device__ float linear(float net) {
	return net;
}

__device__ float linearDerivative(float out) {
	return 1;
}

#pragma endregion

__global__ void multplyMatrices(float* A, float* B, float* C, int widthA, int widthB, int heightA, int heightB) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	float* pA = &A[idx / widthB];
	float* pB = &B[idx % widthB];

	if (idx < widthA * heightB) {
		float out = 0;
		for (int i = 0; i < heightB; i++) {
			out += *pA * *pB;
			pA++;
			pB += widthB;
		}

		C[idx] = out;
	}
}

__global__ void activateMatrices(float* A, int widthA, int heightA, uint8_t activatioonFunction) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx <= widthA * heightA) {
		switch (activatioonFunction) {
		case SIGMOID_FUNCTION:
			A[idx] = sigmoid(A[idx]);
			break;
		case STEP_FUNCTION:
			A[idx] = step(A[idx]);
			break;
		case LINEAR_FUNCTION:
			A[idx] = linear(A[idx]);
			break;
		}
	}
}

__global__ void substractMatrices(float* A, float* B, float* C, int widthA, int heightA) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx <= widthA * heightA) {
		C[idx] = A[idx] - B[idx];
	}
}

__global__ void addMatrices(float* A, float* B, float* C, int widthA, int heightA) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx <= widthA * heightA) {
		C[idx] = A[idx] + B[idx];
	}
}

__global__ void multiplyWithDerivate(float* outputs, float* errorSignal, int sizeOutputs, uint8_t activatioonFunction) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < sizeOutputs) {
		switch (activatioonFunction) {
		case SIGMOID_FUNCTION:
			errorSignal[idx] *= sigmoidDerivative(outputs[idx]);
			break;
		case STEP_FUNCTION:
			errorSignal[idx] *= stepDerivative(outputs[idx]);
			break;
		case LINEAR_FUNCTION:
			errorSignal[idx] *= linearDerivative(outputs[idx]);
			break;
		}
	}
}

__global__ void calculateNewWeights(float* weights, float* Error, float* Inputs, int sizeError, int sizeInputs, float learnRate) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < sizeError * sizeInputs) {
		weights[idx] += -learnRate * Error[idx / sizeInputs] * Inputs[idx % sizeInputs];
	}
}

__global__ void multiplyAndSum(float* weights, float* previousErrorSignal, float* errorSignal, int sizeError, int sizePreviousError) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < sizeError) {
		float sum = 0;
		for (int i = 0; i < sizePreviousError; i++) {
			sum += weights[sizeError * i + idx] * previousErrorSignal[i];
		}
		errorSignal[idx] = sum;
	}
}

int Matrix::copyMatrixToDevice(int leftTopX, int leftTopY, int bottomRightX, int bottomRightY) {
	//printf("Device TopX: %d TopY %d BottomX: %d BottomY %d\n", leftTopX, leftTopY, bottomRightX, bottomRightY);
	int newWidth = bottomRightX - leftTopX + 1;
	int newHeight = bottomRightY - leftTopY + 1;

	int error = hipSuccess;
	hipFree(dataDevice);
	error = hipMalloc(&dataDevice, width * height * sizeof(float));
	if (error != hipSuccess) {
		printf("error while allocating\n");
		return error;
	}
	
	for (int i = leftTopY; i <= bottomRightY; i++) {
		//printf(" i: %d %d %d %d %d\n", i, newWidth * (i - leftTopY), i * width + leftTopX, newWidth, newWidth * newHeight);
		error = hipMemcpy(&dataDevice[newWidth * (i - leftTopY)], &dataHost[i * width + leftTopX], newWidth * sizeof(float), hipMemcpyHostToDevice);
		
		if (error != hipSuccess) {
			printf("error while copying\n");
			return error;
		}
	}
	return error;
}

int Matrix::copyMatrixToHost(int leftTopX, int leftTopY, int bottomRightX, int bottomRightY) {
	//printf("Host TopX: % d TopY % d BottomX : % d BottomY % d\n", leftTopX, leftTopY, bottomRightX, bottomRightY);
	int newWidth = bottomRightX - leftTopX + 1;
	int newHeight = bottomRightY - leftTopY + 1;

	int error = hipSuccess;
	for (int i = leftTopY; i <= bottomRightY; i++) {
		//printf(" i: %d %d %d %d %d\n", i, newWidth * (i - leftTopY), i * width + leftTopX, newWidth, newWidth * newHeight);
		error = hipMemcpy(&dataHost[i * width + leftTopX], &dataDevice[newWidth * (i - leftTopY)], (newWidth) * sizeof(float), hipMemcpyDeviceToHost);
		if (error != hipSuccess) {
			printf("error while copying\n");
			return error;
		}
	}
	return error;
}

int Matrix::copyMatrixToDeviceVector(int top, int bottom) {
	int error;
	hipFree(dataDevice);
	error = hipMalloc(&dataDevice, (bottom - top + 1) * sizeof(float));
	if (error != hipSuccess) {
		printf("error while allocating\n");
		return error;
	}

	error = hipMemcpy(dataDevice, &dataHost[top], (bottom - top + 1) * sizeof(float), hipMemcpyHostToDevice);
	return error;
}

int Matrix::copyMatrixToHostVector(int top, int bottom) {
	int error;

	error = hipMemcpy(&dataHost[top], dataDevice, (bottom - top + 1) * sizeof(float), hipMemcpyDeviceToHost);
	return error;
}

Matrix::Matrix() {

}

Matrix::Matrix(int width, int height, int* errorStatus, uint8_t method, float seed) : height(height), width(width) {
	*errorStatus = hipSuccess;

	dataHost = (float*)malloc(sizeof(float) * height * width);
	int error = hipMalloc(&dataDevice, sizeof(float) * width * height);
	if (error != hipSuccess) {
		printf("Error while allocating matrix-space");
		fflush(stdout);
		*errorStatus = error;
		return;
	}

	if (method == 1) {
		srand(time(0));
		for (int i = 0; i < height; i++) {
			for (int j = 0; j < width; j++) {
				dataHost[i * width + j] = ((float)rand() / (float)rand());
				dataHost[i * width + j] -= (int)dataHost[i * width + j];
			}
		}
	}
	else if (method == 2)
	{
		for (int i = 0; i < height; i++) {
			for (int j = 0; j < width; j++) {
				dataHost[i * width + j] = seed;
			}
		}
	}
}

void Matrix::deleteMatrix() {
	free(dataHost);
	hipFree(dataDevice);
}

unsigned int Matrix::getWidth() {
	return width;
}

unsigned int Matrix::getHeight() {
	return height;
}

float Matrix::getData(unsigned int x, unsigned int y) {
	if (x > width || y > height) return 0;
	return dataHost[y * width + x];
}

void Matrix::setData(unsigned int x, unsigned int y, float value) {
	if (x > width || y > height) return;
	dataHost[y * width + x] = value;
}

void Matrix::printMatrix() {
	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			printf("%+9f ", dataHost[i * width + j]);
		}
		printf("\n");
	}
}

Matrix Matrix::operator * (Matrix B) {
	if (this->width != B.getHeight()) {
		int error;
		return Matrix(0, 0, &error);
		printf("Error ocurred in Matrixmultiplication\n");
	}

	int error = hipSuccess;
	Matrix result = Matrix(B.getWidth(), this->getHeight(), &error, 0);

	long ticker = clock();

	this->copyMatrixToDevice(0, 0, this->getWidth() - 1, this->getHeight());
	B.copyMatrixToDevice(0, 0, B.getWidth() - 1, B.getHeight());

	multplyMatrices <<<B.getWidth() * this->getHeight() / BLOCK_SIZE + 1, BLOCK_SIZE >>> (this->dataDevice, B.dataDevice, result.dataDevice, this->getWidth(), B.getWidth(), this->getHeight(), B.getHeight());
	activateMatrices <<<B.getWidth() * this->getHeight() / BLOCK_SIZE + 1, BLOCK_SIZE >>> (result.dataDevice, result.getWidth(), result.getHeight(), SIGMOID_FUNCTION);

	result.copyMatrixToHost(0, 0, result.getWidth() - 1, result.getHeight());

	ticker = clock() - ticker;
	std::cout << "Ticker: " << ticker << std::endl;

	return result;
}

Matrix Matrix::operator - (Matrix B) {
	if (this->width != B.getWidth() || this->height != B.getHeight()) {
		int error;
		return Matrix(0, 0, &error);
		printf("Error ocurred\n");
	}

	int error = hipSuccess;
	Matrix result = Matrix(B.getWidth(), this->getHeight(), &error, 0);

	long ticker = clock();

	this->copyMatrixToDevice(0, 0, this->getWidth() - 1, this->getHeight());
	B.copyMatrixToDevice(0, 0, B.getWidth() - 1, B.getHeight());

	substractMatrices <<<B.getWidth() * this->getHeight() / BLOCK_SIZE + 1, BLOCK_SIZE >>> (this->dataDevice, B.dataDevice, result.dataDevice, this->getWidth(), this->getHeight());

	result.copyMatrixToHost(0, 0, result.getWidth() - 1, result.getHeight());

	return result;
}

Matrix Matrix::operator + (Matrix B) {
	if (this->width != B.getWidth() || this->height != B.getHeight()) {
		int error;
		return Matrix(0, 0, &error);
		printf("Error ocurred in Matrixaddition\n");
	}

	int error = hipSuccess;
	Matrix result = Matrix(B.getWidth(), this->getHeight(), &error, 0);

	long ticker = clock();

	this->copyMatrixToDevice(0, 0, this->getWidth() - 1, this->getHeight());
	B.copyMatrixToDevice(0, 0, B.getWidth() - 1, B.getHeight());

	addMatrices << <B.getWidth() * this->getHeight() / BLOCK_SIZE + 1, BLOCK_SIZE >> > (this->dataDevice, B.dataDevice, result.dataDevice, this->getWidth(), this->getHeight());

	result.copyMatrixToHost(0, 0, result.getWidth() - 1, result.getHeight());

	return result;
}

void Matrix::multiplyWithDerivateMatrix(Matrix* errorSignal, int activationFunction) {
	if(this->height != errorSignal->getHeight() + 1) {
		int error;
		printf("Error ocurred in multiply with Derivate\n");
		printf("Output %d\tError%d", this->height, errorSignal->getHeight());
	}

	int error = hipSuccess;

	/*printf("Eingabe\n");
	this->printMatrix();*/

	this->copyMatrixToDeviceVector(0, this->getHeight() - 2);
	errorSignal->copyMatrixToDeviceVector(0, errorSignal->getHeight() - 1);

	multiplyWithDerivate <<<errorSignal->getWidth() * this->getHeight() - 1 / BLOCK_SIZE + 1, BLOCK_SIZE >>> (this->dataDevice, errorSignal->dataDevice, this->getHeight(), activationFunction);

	errorSignal->copyMatrixToHostVector(0, errorSignal->getHeight() - 1);
}

void Matrix::calculateNewWeightsMatrix(Matrix* Inputs, Matrix* Error, float learnRate) {
	if (this->height != Error->getHeight() || this->width != Inputs->getHeight()) {
		int error;
		printf("Error ocurred in calculated Weights\n");
	}

	int error = hipSuccess;

	Error->copyMatrixToDeviceVector(0, Error->getHeight() - 1);
	Inputs->copyMatrixToDeviceVector(0, Inputs->getHeight() - 1);
	/*printf("\n");
	Error->printMatrix();
	printf("\n");
	Inputs->printMatrix();
	printf("\n");
	this->printMatrix();*/


	calculateNewWeights <<<this->getWidth() * this->getHeight() / BLOCK_SIZE + 1, BLOCK_SIZE >>> (this->dataDevice, Error->dataDevice, Inputs->dataDevice, this->getHeight(), this->getWidth(), learnRate);

	this->copyMatrixToHost(0, 0, this->getWidth() - 1, this->getHeight() - 1);
}

void Matrix::multiplyAndSumMatrix(Matrix* weights, Matrix* previousErrorSignal) {
	if (this->height + 1 != weights->getWidth() || weights->getHeight() != previousErrorSignal->getHeight()) {
		int error;
		printf("Error ocurred in Multiply and Sum heightPrevious %d heightError %d widthWeights %d heightWeights %d\n", previousErrorSignal->height, this->height, weights->width, weights->height);
	}

	int error = hipSuccess;

	weights->copyMatrixToDevice(0, 0, weights->getWidth() - 2, weights->getHeight() - 1);
	previousErrorSignal->copyMatrixToDeviceVector(0, previousErrorSignal->getHeight() - 1);

	multiplyAndSum <<<this->getWidth() * this->getHeight() / BLOCK_SIZE + 1, BLOCK_SIZE >>> (weights->dataDevice, previousErrorSignal->dataDevice, this->dataDevice, this->getHeight(), previousErrorSignal->getHeight());

	this->copyMatrixToHostVector(0, this->getHeight() - 1);
}

void Matrix::Forward(Matrix& Inputs, Matrix& Weights, uint8_t activationFunction) {
	if (Weights.width != Inputs.getHeight()) {
		int error;
		printf("Error ocurred in Matrixmultiplication WeightsWidth %d InputsHeight %d\n", Weights.getWidth(), Inputs.getHeight());
		return;
	}
	//printf("WeightsWidth %d AHeight %d\n", Weights.getWidth(), Inputs.getHeight());

	Inputs.copyMatrixToDeviceVector(0, Inputs.getHeight() - 1);
	Weights.copyMatrixToDevice(0, 0, Weights.getWidth() - 1, Weights.getHeight() - 1);

	multplyMatrices <<<Weights.getHeight() * Inputs.getWidth() / BLOCK_SIZE + 1, BLOCK_SIZE >>> (Weights.dataDevice, Inputs.dataDevice, this->dataDevice, Weights.getWidth(), Inputs.getWidth(), Weights.getHeight(), Inputs.getHeight());
	activateMatrices <<<Weights.getHeight() * Inputs.getWidth() / BLOCK_SIZE + 1, BLOCK_SIZE >>> (this->dataDevice, this->getWidth(), this->getHeight() - 1, activationFunction);

	this->copyMatrixToHostVector(0, this->getHeight() - 2);
}

void Matrix::SubstactTargetFromOutput(Matrix& A, Matrix& B){
	if(A.getHeight() - 1 != B.getHeight()){
		int error;
		printf("Error ocurred in Matrixmultiplication A Height %d B Height %d\n", A.getHeight(), B.getHeight());
		return;
	}

	A.copyMatrixToDevice(0, 0, A.getWidth() - 1, A.getHeight() - 1);
	B.copyMatrixToDevice(0, 0, B.getWidth() - 1, B.getHeight() - 1);

	substractMatrices <<<B.getWidth() * A.getHeight() / BLOCK_SIZE + 1, BLOCK_SIZE >>> (A.dataDevice, B.dataDevice, this->dataDevice, this->getWidth(), this->getHeight());

	this->copyMatrixToHost(0, 0, this->getWidth() - 1, this->getHeight() - 1);
}