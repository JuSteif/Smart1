#include "Layer.h"

Layer::Layer(int sizeInput, int sizeOutput, bool outputNeuron, Matrix* Inputs, Matrix* PreviousWeights, Matrix* PreviousErrorSignal) : outputNeuron(outputNeuron), Inputs(Inputs), PreviousWeights(PreviousWeights), PreviousErrorSignal(PreviousErrorSignal) {
	int error;
	Weights = Matrix(sizeInput, sizeOutput, &error, 2, 1);
	if (error != hipSuccess) return;
	Outputs = Matrix(1, sizeOutput + 1, &error, 0);
	if (error != hipSuccess) return;
	Outputs.setData(Outputs.getWidth() - 1, Outputs.getHeight() - 1, 1);
	ErrorSignal = Matrix(1, sizeOutput, &error, 0);
	if (error != hipSuccess) return;
}

void Layer::deleteLayer() {
	Weights.deleteMatrix();
	Outputs.deleteMatrix();
}

void Layer::forward() {
	//Outputs = Weights * *Inputs;
	Outputs.Forward(*Inputs, Weights, SIGMOID_FUNCTION);
}

void Layer::calculatErrorSignal() {
	//*
	printf("\nPreviousErrorSignalFunction:\n");
	PreviousErrorSignal->printMatrix();

	if (outputNeuron) {
		//ErrorSignal = Outputs - *PreviousErrorSignal;
		ErrorSignal.SubstactTargetFromOutput(Outputs, *PreviousErrorSignal);
		printf("Last Layer\n");
	}
	else {
		printf("\nPreviousWeights:\n");
		PreviousWeights->printMatrix();

		ErrorSignal.multiplyAndSumMatrix(PreviousWeights, PreviousErrorSignal);
		//ErrorSignal = *PreviousWeights * *PreviousErrorSignal;
		printf("Hidden Layer\n");
	}
	printf("Error Signal\n");
	ErrorSignal.printMatrix();
	printf("\nErrorSignalFunctionAfter:\n");
	ErrorSignal.printMatrix();

	Outputs.multiplyWithDerivateMatrix(&ErrorSignal, SIGMOID_FUNCTION);
}

void Layer::calculateNewWeights(float learnRate) {
	Weights.calculateNewWeightsMatrix(Inputs, &ErrorSignal, learnRate);
}