#include "Layer.h"

Layer::Layer(int sizeInput, int sizeOutput, bool outputNeuron, Matrix* Inputs, Matrix* PreviousWeights, Matrix* PreviousErrorSignal) : outputNeuron(outputNeuron), Inputs(Inputs), PreviousWeights(PreviousWeights), PreviousErrorSignal(PreviousErrorSignal) {
	int error;
	Weights = Matrix(sizeInput, sizeOutput, &error, 2, 1);
	if (error != hipSuccess) return;
	Outputs = Matrix(1, sizeOutput, &error, 0);
	if (error != hipSuccess) return;
	ErrorSignal = Matrix(1, sizeOutput, &error, 0);
	if (error != hipSuccess) return;
}

void Layer::deleteLayer() {
	Weights.deleteMatrix();
	Outputs.deleteMatrix();
}

void Layer::forward() {
	Outputs = Weights * *Inputs;
}

void Layer::calculatErrorSignal() {
	//*
	printf("\nPreviousErrorSignalFunction:\n");
	PreviousErrorSignal->printMatrix();
	/*printf("\nOutputs:\n");
	Outputs.printMatrix();
	//*/

	if (outputNeuron) {
		ErrorSignal = Outputs - *PreviousErrorSignal;
		printf("Last Layer\n");
	}
	else {
		printf("\nPreviousWeights:\n");
		PreviousWeights->printMatrix();

		ErrorSignal.multiplyAndSumMatrix(PreviousWeights, PreviousErrorSignal);
		//ErrorSignal = *PreviousWeights * *PreviousErrorSignal;
		printf("Hidden Layer\n");
	}
	//*
	printf("\nErrorSignalFunctionAfter:\n");
	ErrorSignal.printMatrix();
	//*/

	Outputs.multiplyWithDerivateMatrix(&ErrorSignal, SIGMOID_FUNCTION);
}

void Layer::calculateNewWeights(float learnRate) {
	Weights.calculateNewWeightsMatrix(Inputs, &ErrorSignal, learnRate);
}